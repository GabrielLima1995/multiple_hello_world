#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


//device function (Kernel) 

__global__ void hello_cuda()
{
	printf("Hello Cuda Team =)\n");

}


//host function 

int main()

{

	dim3 grid(2, 2, 1);
	dim3 block(8, 2, 1);

	hello_cuda << <grid,block >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();

	return 0;
}